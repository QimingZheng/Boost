#include "hip/hip_runtime.h"
#include<stdio.h>
#include"hip/hip_runtime_api.h"

const int SIZE = 35 * 1024;
const int ThreadPerBlock = 256;
const int BlockPerGrid = min( 32, (SIZE + ThreadPerBlock -1)/ThreadPerBlock);

__global__ void dot(float *a, float *b, float *p_c)
{
    __shared__ float cache[ThreadPerBlock];
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    float tmp = 0.0;
    while(tid < SIZE){
        tmp += a[tid]*b[tid];
        tid += blockDim.x*gridDim.x;
    }

    cache[threadIdx.x] = tmp;

    __syncthreads();

    int i = blockDim.x/2;
    while(i){
        if(threadIdx.x < i){
            cache[threadIdx.x] += cache[threadIdx.x+i];
        }
        i >>= 1;
        __syncthreads();
    }
    if (threadIdx.x==0){
        p_c[blockIdx.x] = cache[0];
    }

}

int main(){
    float *h_a, *h_b, h_c, *h_partial_c;
    float *d_a, *d_b, *d_partial_c;
    h_a = (float *)malloc(SIZE*sizeof(float));
    h_b = (float *)malloc(SIZE*sizeof(float));
    h_partial_c = (float*)malloc(BlockPerGrid*sizeof(float));

    checkCudaErrors(hipMalloc((void**)&d_a, SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_b, SIZE * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_partial_c, BlockPerGrid * sizeof(float)));

    for(int i=0;i<SIZE;i++)
    {
    h_a[i] = i;
    h_b[i] = 3*i;
    }

    checkCudaErrors(hipMemcpy(d_a, h_a, SIZE * sizeof(float), hipMemcpyHostToDevice)); 
    checkCudaErrors(hipMemcpy(d_b, h_b, SIZE * sizeof(float), hipMemcpyHostToDevice)); 
    checkCudaErrors(hipMemcpy(d_partial_c, h_partial_c, BlockPerGrid * sizeof(float), hipMemcpyHostToDevice)); 

    dot<<<BlockPerGrid, ThreadPerBlock>>>(d_a,d_b,d_partial_c);

    checkCudaErrors(hipMemcpy(h_partial_c, d_partial_c, BlockPerGrid * sizeof(float), hipMemcpyDeviceToHost)); 
    h_c = 0.0;
    for(int i=0;i<BlockPerGrid;i++)
    h_c += h_partial_c[i];
    
    printf("Sum = %.6f \n", h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_partial_c);

    free(h_a);
    free(h_b);
    free(h_partial_c);

    return 0;
}
