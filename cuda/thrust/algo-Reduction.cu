#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>

// square<T> computes the square of a number f(x) -> x*x
template <typename T>
struct square
{
  __host__ __device__
  T operator()(const T& x) const
  { 
    return x * x;
  }
};

int main(void)
{
  // initialize host array
  float x[4] = {1.0, 2.0, 3.0, 4.0};

  // transfer to device
  thrust::device_vector<float> d_x(x, x + 4); //利用x来初始化d_x

  // setup arguments
  square<float>        unary_op;   //一元运算
  thrust::plus<float> binary_op;  //二元运算
  float init = 0;

  // compute norm
  float norm = std::sqrt( thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op) );

  std::cout << norm << std::endl;

  return 0;
}
